#include "hip/hip_runtime.h"
#include "cudapca.h"

#include <cmath>
#include <cstdio>
#include <cstdlib>

#include "hipblas.h"

#include <thrust/device_vector.h>

// Auxiliary functions

#ifdef CUDAPCA_USE_FLOAT
#define cublasXaxpy hipblasSaxpy
#define cublasXdot hipblasSdot
#define cublasXgeam hipblasSgeam
#define cublasXgemm hipblasSgemm
#define cublasXgemv hipblasSgemv
#define cublasXger hipblasSger
#define cublasXnrm2 hipblasSnrm2
#define cublasXscal hipblasSscal
#else
#define cublasXaxpy hipblasDaxpy
#define cublasXdot hipblasDdot
#define cublasXgeam hipblasDgeam
#define cublasXgemm hipblasDgemm
#define cublasXgemv hipblasDgemv
#define cublasXger hipblasDger
#define cublasXnrm2 hipblasDnrm2
#define cublasXscal hipblasDscal
#endif


#define cudaCheck(call)                                                        \
{                                                                              \
    const hipError_t stat = call;                                             \
    if (stat != hipSuccess) {                                                 \
        fprintf(stderr, "Error %s at line %d in file %s\n",                    \
                hipGetErrorString(stat), __LINE__, __FILE__);                 \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}


#define cublasCheck(call)                                                      \
{                                                                              \
    const hipblasStatus_t stat = call;                                          \
    if (stat != HIPBLAS_STATUS_SUCCESS) {                                       \
        fprintf(stderr, "CUBLAS error %d at line %d in file %s\n",             \
                stat, __LINE__, __FILE__);                                     \
        exit(EXIT_FAILURE);                                                    \
    }                                                                          \
}


inline int
ceilDiv(const int a, const int b)
{
    return ((a + b - 1) / b);
}


// CUDA kernels

__global__ void
kernGeneratePatches(
        CUDAPCA::data_t *const g_patchData,
        const CUDAPCA::data_t *const g_data,
        const int depth, const int height, const int width,
        const int patchRadius)
{
    // Get and check thread index
    const int dataSize = depth * height * width;
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= dataSize) return;

    // Get position in the volume of the center of the patch
    const int ck = idx / (width * height);
    const int cj = (idx - ck * width * height) / width;
    const int ci = idx % width;

    // Build patch. Values outside edges are nullified
    const int patchDiam = (2 * patchRadius + 1);
    const int patchSize = patchDiam * patchDiam * patchDiam;
    CUDAPCA::data_t *const g_patchDataLocal = g_patchData + idx * patchSize;
    int cont = 0;

    for (int k = -patchRadius; k <= +patchRadius; k++) {
        for (int j = -patchRadius; j <= +patchRadius; j++) {
            for (int i = -patchRadius; i <= +patchRadius; i++) {
                const int pk = ck + k;
                const int pj = cj + j;
                const int pi = ci + i;

                if (0 <= pk && pk < depth
                        && 0 <= pj && pj < height
                        && 0 <= pi && pi < width) {

                    const int pos = pi + pj * width + pk * width * height;
                    g_patchDataLocal[cont++] = g_data[pos];
                } else {
                    g_patchDataLocal[cont++] = 0;
                }
            }
        }
    }
}


// Class CUDAPCAData

CUDAPCA::CUDAPCAData::CUDAPCAData(
        const int _depth,
        const int _height,
        const int _width,
        const data_t *const _data)
    : depth(_depth)
    , height(_height)
    , width(_width)
    , data(0)
{
    init(depth * height * width, _data);
}


CUDAPCA::CUDAPCAData::CUDAPCAData(
        const int _depth,
        const int _height,
        const int _width,
        const size_t dataSize,
        const data_t *const _data)
    : depth(_depth)
    , height(_height)
    , width(_width)
    , data(0)
{
    init(dataSize, _data);
}


CUDAPCA::CUDAPCAData::~CUDAPCAData() {
    printf("Releasing GPU memory...");
    cudaCheck(hipFree(const_cast<data_t *>(data)));
}


void
CUDAPCA::CUDAPCAData::init(
        const size_t dataSize,
        const data_t *const _data)
{
    cudaCheck(hipMalloc(reinterpret_cast<void **>(const_cast<data_t **>(&data)),
                         dataSize * sizeof(*data)));

    cudaCheck(hipMemcpy(const_cast<data_t *>(data), _data,
                         dataSize * sizeof(*data),
                         hipMemcpyDeviceToDevice));
}


// Class CUDAPCAPatches

CUDAPCA::CUDAPCAPatches::CUDAPCAPatches(
        const int _depth,
        const int _height,
        const int _width,
        const int _patchRadius,
        const data_t *const _data)
    : CUDAPCAData(_depth, _height, _width,
                  _depth * _height * _width * std::pow(2*_patchRadius+1, 3),
                  _data)
    , patchRadius(_patchRadius)
{

}


// Namespace functions

CUDAPCA::CUDAPCAData
CUDAPCA::uploadData(
        const int depth,
        const int height,
        const int width,
        const data_t *const h_data)
{
    const int dataSize = depth * height * width;
    data_t *d_data;

    // Allocate and copy data to GPU memory
    cudaCheck(hipMalloc(reinterpret_cast<void**>(&d_data),
                         dataSize * sizeof(*d_data)));

    cudaCheck(hipMemcpy(d_data, h_data,
                         dataSize * sizeof(*d_data),
                         hipMemcpyHostToDevice));

    // Generate CUDAPCAData object, clean and return
    CUDAPCAData h_pcaData(depth, width, height, d_data);

    cudaCheck(hipFree(d_data));

    return h_pcaData;
}


std::auto_ptr<CUDAPCA::data_t>
CUDAPCA::downloadData(const CUDAPCA::CUDAPCAData &d_data)
{
    const int dataSize = d_data.depth * d_data.height * d_data.width;
    data_t *h_data;

    // Allocate space in host and copy data from GPU memory
    cudaCheck(hipHostMalloc(reinterpret_cast<void**>(&h_data),
                             dataSize * sizeof(*h_data)));

    cudaCheck(hipMemcpy(h_data, d_data.data,
                         dataSize * sizeof(*h_data),
                         hipMemcpyDeviceToHost));

    return std::auto_ptr<data_t>(h_data);
}


CUDAPCA::CUDAPCAPatches
CUDAPCA::generatePatches(
        const CUDAPCA::CUDAPCAData &d_data,
        const int patchRadius)
{
    // Allocate patch space in GPU memory
    const int patchDiam = 2 * patchRadius + 1;
    const int patchSize = patchDiam * patchDiam * patchDiam;
    const int dataSize = d_data.depth * d_data.height * d_data.width;

    data_t *d_patchData;
    cudaCheck(hipMalloc((void **) &d_patchData,
                               patchSize * dataSize * sizeof(*d_patchData)));

    // Launch kernel and wait to finish
    const dim3 dimBlk(512);
    const dim3 dimGrd(ceilDiv(dataSize, dimBlk.x));

    kernGeneratePatches<<<dimGrd, dimBlk>>>(
            d_patchData, d_data.data,
            d_data.depth, d_data.width, d_data.height,
            patchRadius);

    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipGetLastError());

    CUDAPCAPatches patches(
            d_data.depth, d_data.width, d_data.height,
            patchRadius, d_patchData);

    cudaCheck(hipFree(d_patchData));

    return patches;
}


std::auto_ptr<CUDAPCA::data_t>
CUDAPCA::downloadPatches(const CUDAPCA::CUDAPCAPatches &d_patches)
{
    const int patchDiam = 2 * d_patches.patchRadius + 1;
    const int patchSize = patchDiam * patchDiam * patchDiam;
    const int patchesSize = d_patches.depth * d_patches.height
                            * d_patches.width * patchSize;
    data_t *h_patches;

    // Allocate space in host and copy data from GPU memory
    cudaCheck(hipHostMalloc((void **) &h_patches,
                             patchesSize * sizeof(*h_patches)));

    cudaCheck(hipMemcpy(h_patches, d_patches.data,
                         patchesSize * sizeof(*h_patches),
                         hipMemcpyDeviceToHost));

    return std::auto_ptr<data_t>(h_patches);
}


CUDAPCA::CUDAPCAData
CUDAPCA::generateEigenvecs(const CUDAPCA::CUDAPCAPatches &d_patches)
{
    // Create CUBLAS handle
    hipblasHandle_t handle;
    cublasCheck(hipblasCreate(&handle));

    // Each patch is considered as a sample of as many random variables as
    // the patch size. Each patch is stored as a row of the d_patches matrix,
    // and each column corresponds to a distinct random variable.
    const int dataSize = d_patches.depth * d_patches.height * d_patches.width;
    const int patchDiam = (2 * d_patches.patchRadius + 1);
    const int patchSize = patchDiam * patchDiam * patchDiam;

    // Compute the mean patch, in several steps:
    //
    // 1.   Multiply the columns of patch matrix space by the one-vector.
    //      For each column/random variable, the sum of the samples is
    //      obtained.
    const thrust::device_vector<data_t> d_ones(dataSize, 1.f);
    data_t alpha = 1.f;
    data_t beta = 0.f;


    // Note: BLAS gemv function computes y = aAx + by. All BLAS functions
    // expect matrices to be stored in column-major format (a la Fortran).
    // A rows correspond to single patches, I do not need transpose the
    // matrix.
    thrust::device_vector<data_t> d_sum(patchSize, 0.f);

    cublasCheck(cublasXgemv(handle, HIPBLAS_OP_N,
                            patchSize, dataSize,
                            &alpha,
                            d_patches.data, patchSize,
                            d_ones.data().get(), 1,
                            &beta,
                            d_sum.data().get(), 1));

//#define TEST_EIGEN_SUM
#ifdef TEST_EIGEN_SUM
    // Get values of sum and return them
    data_t *d_sum_copy;

    cudaCheck(hipMalloc((void**) &d_sum_copy,
                         patchSize * sizeof(*d_sum_copy)));

    cudaCheck(hipMemcpy(d_sum_copy, d_sum.data().get(),
                         patchSize * sizeof(*d_sum_copy),
                         hipMemcpyHostToDevice));

    return CUDAPCAData(1, 1, patchSize, d_sum_copy);
#endif

    // 2.   Divide the sum vector between the number of samples to
    //      get the mean patch. BLAS scal function computes x = ax.
    thrust::device_vector<data_t> d_mean(d_sum);
    alpha = 1.f / dataSize;

    cublasCheck(cublasXscal(handle, patchSize, &alpha,
                            d_mean.data().get(), 1));

//#define TEST_EIGEN_MEAN
#ifdef TEST_EIGEN_MEAN
    // Get values of mean and return them
    data_t *d_mean_copy;

    cudaCheck(hipMalloc((void**) &d_mean_copy,
                         patchSize * sizeof(*d_mean_copy)));

    cudaCheck(hipMemcpy(d_mean_copy, d_mean.data().get(),
                         patchSize * sizeof(*d_mean_copy),
                         hipMemcpyHostToDevice));

    return CUDAPCAData(1, 1, patchSize, d_mean_copy);
#endif

    // Compute all possible binary products between the means of all the
    // random variables (i.e., 2^(patch size) possible products, ignoring
    // the commutative property). The resulting matrix contains in each
    // cell (i, j) the value E(Xi) * E(Xj). The BLAS function ger computes
    // A = axx^T + A. I use this function instead of spr because the
    // latter returns a packed matrix, and I want a full-fledged one.
    thrust::device_vector<data_t> d_prodmean(patchSize * patchSize, 0.f);
    alpha = 1.f;

    cublasCheck(cublasXger(handle, patchSize, patchSize,
                           &alpha,
                           d_mean.data().get(), 1,
                           d_mean.data().get(), 1,
                           d_prodmean.data().get(), patchSize));

//#define TEST_EIGEN_PRODMEAN
#ifdef TEST_EIGEN_PRODMEAN
    // Get values of mean product and return them
    data_t *d_prodmean_copy;

    cudaCheck(hipMalloc((void**) &d_prodmean_copy,
                         patchSize * patchSize * sizeof(*d_prodmean_copy)));

    cudaCheck(hipMemcpy(d_prodmean_copy, d_prodmean.data().get(),
                         patchSize * patchSize * sizeof(*d_prodmean_copy),
                         hipMemcpyHostToDevice));

    return CUDAPCAData(1, patchSize, patchSize, d_prodmean_copy);
#endif

    // Compute the mean of all possible binary products of all the random
    // products (i.e., 2^(patch size) possible products, ignoring the
    // commutative property). The resulting matrix should contain in each
    // cell (i, j) the value E(Xi * Xj). This is done in several steps:
    //
    // 1.   Compute a matrix where each cell (i,j) is the sum of the
    //      products of the elements i and j of all patches. The resulting
    //      matrix contains in each cell (i, j) the value of sum(Xi * Xj).
    //      This is accomplished by multiplying the transposed patch
    //      matrix by itself (or the other way round in column-major
    //      format). BLAS gemm function computes C = aAB + bC.
    thrust::device_vector<data_t> d_sumprod(patchSize * patchSize);
    alpha = 1.f;
    beta = 0.f;

    cublasCheck(cublasXgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                            patchSize, patchSize, dataSize,
                            &alpha,
                            d_patches.data, patchSize,
                            d_patches.data, patchSize,
                            &beta,
                            d_sumprod.data().get(), patchSize));

//#define TEST_EIGEN_SUMPROD
#ifdef TEST_EIGEN_SUMPROD
    // Get values of product and return them
    data_t *d_sumprod_copy;

    cudaCheck(hipMalloc((void**) &d_sumprod_copy,
                         patchSize * patchSize * sizeof(*d_sumprod_copy)));

    cudaCheck(hipMemcpy(d_sumprod_copy, d_sumprod.data().get(),
                         patchSize * patchSize * sizeof(*d_sumprod_copy),
                         hipMemcpyHostToDevice));

    return CUDAPCAData(1, patchSize, patchSize, d_sumprod_copy);
#endif

    // 2.   Divide the sum vector between the number of samples to
    //      get the mean of all products.
    thrust::device_vector<data_t> d_meanprod(d_sumprod);
    alpha = 1.f / dataSize;

    cublasCheck(cublasXscal(handle, patchSize * patchSize, &alpha,
                            d_meanprod.data().get(), 1));

//#define TEST_EIGEN_MEANPROD
#ifdef TEST_EIGEN_MEANPROD
    // Get values of mean product and return them
    data_t *d_meanprod_copy;

    cudaCheck(hipMalloc((void**) &d_meanprod_copy,
                         patchSize * patchSize * sizeof(*d_meanprod_copy)));

    cudaCheck(hipMemcpy(d_meanprod_copy, d_meanprod.data().get(),
                         patchSize * patchSize * sizeof(*d_meanprod_copy),
                         hipMemcpyHostToDevice));

    return CUDAPCAData(1, patchSize, patchSize, d_meanprod_copy);
#endif

    // Compute the covariance matrix. This matrix contains all possible
    // covariance values for all the random variables. That is, cell
    // (i,j) contains the following value:
    //
    //              Cov(Xi, Xj) = E(Xi * Xj) - E(Xi) * E(Xj)
    //
    // This matrix is obtained from the results computed above. BLAS
    // famous function axpy computes y = ax + y.
    thrust::device_vector<data_t> d_cov(d_meanprod);
    alpha = -1.f;

    cublasCheck(cublasXaxpy(handle, patchSize * patchSize, &alpha,
                            d_prodmean.data().get(), 1,
                            d_cov.data().get(), 1));

//#define TEST_EIGEN_COV
#ifdef TEST_EIGEN_COV
    // Get values of covariance and return them
    data_t *d_cov_copy;

    cudaCheck(hipMalloc((void**) &d_cov_copy,
            patchSize * patchSize * sizeof(*d_cov_copy)));

    cudaCheck(hipMemcpy(d_cov_copy, d_cov.data().get(),
            patchSize * patchSize * sizeof(*d_cov_copy),
            hipMemcpyHostToDevice));

    return CUDAPCAData(1, patchSize, patchSize, d_cov_copy);
#endif

    // Compute eigenvectors from the covariance matrix. The approach used
    // here is an iterative one. It is based on the code of Adams et al.
    // "Gaussian KD-Trees for Fast High-Dimensional Filtering" (2009).
    // Consider each row in the covariance matrix a first approximation
    // of each eigenvector. The code proceeds iteratively in several steps.
    thrust::device_vector<data_t> d_eigenvecs(d_cov);
    thrust::device_vector<data_t> d_eigenvecs_old(patchSize * patchSize, 0.f);

    while(true) {
        // 1. Find an orthogonal base for the current eigenvectors
        for (int i = 0; i < patchSize; i++) {
            // Make vector i independent of all previous ones
            for (int j = 0; j < i; j++) {
                // Compute dot = v_i * v_j, where * is the dot product
                data_t dot = 0.f;
                cublasCheck(cublasXdot(handle, patchSize,
                                       d_eigenvecs.data().get() + i*patchSize, 1,
                                       d_eigenvecs.data().get() + j*patchSize, 1,
                                       &dot));

                // Compute v_i = -dot * v_j + v_i
                dot = -dot;
                cublasCheck(cublasXaxpy(handle, patchSize, &dot,
                                        d_eigenvecs.data().get() + j*patchSize, 1,
                                        d_eigenvecs.data().get() + i*patchSize, 1));
            }

            // Normalize vector
            // Compute norm = norm(v_i)
            data_t norm = 0.f;
            cublasCheck(cublasXnrm2(handle, patchSize,
                                    d_eigenvecs.data().get() + i*patchSize, 1,
                                    &norm));


            // Compute v_i = v_i / norm
            norm = 1.f / norm;

            cublasCheck(cublasXscal(handle, patchSize, &norm,
                                    d_eigenvecs.data().get() + i*patchSize, 1))
        }

        /// TODO Test if previous step works
        // 2. Check convergence
        // Compute d_diff = d_eigen - d_old
        thrust::device_vector<data_t> d_diff(patchSize * patchSize);

        alpha = 1.0f;
        beta = -1.0f;
        cublasCheck(cublasXgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                patchSize, patchSize,
                                &alpha,
                                d_eigenvecs.data().get(), patchSize,
                                &beta,
                                d_eigenvecs_old.data().get(), patchSize,
                                d_diff.data().get(), patchSize));

        // Accumulate norm of the difference vectors
        data_t dist = 0.f;
        for (int i = 0; i < patchSize; i++) {
            data_t norm = 0.f;
            cublasCheck(cublasXnrm2(handle, patchSize,
                                    d_diff.data().get() + i*patchSize, 1,
                                    &norm));

            dist += norm;
        }

        // Break loop if solution has converged
        printf("dist=%12.6f\n", dist);
        if (dist < 0.001f) break;

        // Solution has not converged, so we keep iterating. Multiply
        // the covariance matrix by current eigenvectors.
        alpha = 1.f;
        beta = 1.f;
        cublasCheck(cublasXgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                patchSize, patchSize, patchSize,
                                &alpha,
                                d_cov.data().get(), patchSize,
                                d_eigenvecs.data().get(), patchSize,
                                &beta,
                                d_eigenvecs_old.data().get(), patchSize));

        // Swap old and current eigenvectors
        d_eigenvecs_old.swap(d_eigenvecs);
    }

#define TEST_EIGEN_COMP
#ifdef TEST_EIGEN_COMP
    // Get values of eigenvectors and return them
    data_t *d_eigenvecs_copy;

    cudaCheck(hipMalloc((void**) &d_eigenvecs_copy,
            patchSize * patchSize * sizeof(*d_eigenvecs_copy)));

    cudaCheck(hipMemcpy(d_eigenvecs_copy, d_eigenvecs.data().get(),
            patchSize * patchSize * sizeof(*d_eigenvecs_copy),
            hipMemcpyHostToDevice));

    return CUDAPCAData(1, patchSize, patchSize, d_eigenvecs_copy);
#endif

    // Clean and return
    cublasCheck(hipblasDestroy(handle));
    return CUDAPCAData(0, 0, 0, 0);
}
